#include "hip/hip_runtime.h"
#include "cuda_select.cuh"
#include <hip/hip_cooperative_groups.h>
#include <vector>

CudaSelect::CudaSelect() {

  m_firstMethodWasOr_b = true;
  m_firstRun_b = true;

  m_resultDatabase_v.clear();
}

CudaSelect::~CudaSelect() {}

__global__ void searcData(long int *f_dataBase_p, long int *f_resultLines_p,
                          const unsigned int f_databaseRowSize_ui,
                          const unsigned int f_databaseColumnSize_ui,
                          const unsigned int f_targetWord_ui) {

  int rowThread = blockIdx.x * blockDim.x + threadIdx.x;
  int columnThread = blockIdx.y * blockDim.y + threadIdx.y;

  long int l_tmpWordContainer_li =
      f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + columnThread];

  if (l_tmpWordContainer_li == f_targetWord_ui) {

    for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {

      long int l_dataBaseFoundedLineContent_li =
          f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];
      f_resultLines_p[l_it_x + (rowThread * f_databaseColumnSize_ui)] =
          l_dataBaseFoundedLineContent_li;
    }
  }
  // auto syncOnlyThreads= cooperative_groups::this_thread();
  // auto syncGroup = cooperative_groups::this_thread_block();
  // syncGroup.sync();
  // syncOnlyThreads.sync();
  __syncthreads();
}

__global__ void searcDataInColumn(long int *f_dataBase_p,
                                  long int *f_resultLines_p,
                                  const unsigned int f_databaseRowSize_ui,
                                  const unsigned int f_databaseColumnSize_ui,
                                  const long int f_targetWord_ui,
                                  const unsigned int f_targetColumn) {

  int rowThread = blockIdx.x * blockDim.x + threadIdx.x;

  // printf("%i \n",rowThread);
  long int l_tmpWordContainer_li =
      f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + f_targetColumn];

  if (l_tmpWordContainer_li == f_targetWord_ui) {

    for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {

      long int l_dataBaseFoundedLineContent_li =
          f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];
      f_resultLines_p[l_it_x + (rowThread * f_databaseColumnSize_ui)] =
          l_dataBaseFoundedLineContent_li;
    }
  }
  // auto syncOnlyThreads= cooperative_groups::this_thread();
  // auto syncGroup = cooperative_groups::this_thread_block();
  // syncGroup.sync();
  // syncOnlyThreads.sync();
  __syncthreads();
}

void CudaSelect::copyDataToDevice(
    const vector<vector<long int>> &f_dataBase_r,
    const unsigned int f_databaseRowSize_ui,
    unsigned int f_databaseColumnSize_ui,
    thrust::device_vector<long int> &f_DeviceDataBase_r) {

  int l_tmpDatabaseContainer_i[f_databaseRowSize_ui][f_databaseColumnSize_ui];

  unsigned int l_it_x = 0;
  unsigned int l_it_y = 0;

  for (vector<long int> vec : f_dataBase_r) {
    for (long int vector_member : vec) {
      l_tmpDatabaseContainer_i[l_it_x][l_it_y] = vector_member;
      l_it_y++;
    }
    l_it_y = 0;
    l_it_x++;
  }

  // ToDo! What??
  thrust::copy(&(l_tmpDatabaseContainer_i[0][0]),
               &(l_tmpDatabaseContainer_i[f_databaseRowSize_ui]
                                         [f_databaseColumnSize_ui]),
               f_DeviceDataBase_r.begin());
}

void CudaSelect::copyDataFromDevice(
    const unsigned int f_databaseRowSize_ui,
    const unsigned int f_databaseColumnSize_ui,
    const thrust::host_vector<long int> &f_resultVector) {

  vector<long int> l_tmpDatabaseContainer_v;
  for (int x = 0; x < f_databaseRowSize_ui; x++) {

    l_tmpDatabaseContainer_v.clear();
    for (int y = 0; y < f_databaseColumnSize_ui; y++) {
      long int l_tmpVectorValu =
          f_resultVector[(x * f_databaseColumnSize_ui) + y];

      l_tmpDatabaseContainer_v.push_back(l_tmpVectorValu);
    }
    m_resultDatabase_v.push_back(l_tmpDatabaseContainer_v);
  }
}

void CudaSelect::CudaRun(const vector<string> &f_selectRule,
                         const vector<vector<long int>> &f_dataBase_r,
                         const vector<string> &f_dataBaseHeader_v) {

  unsigned int l_databaseRowSize_ui = f_dataBase_r.size();
  unsigned int l_databaseColumnSize_ui = f_dataBaseHeader_v.size();

  thrust::device_vector<long int> *l_collectDataVector_p(NULL);
  thrust::device_vector<long int> l_workDataVector(l_databaseRowSize_ui *
                                                   l_databaseColumnSize_ui);

  thrust::device_vector<long int> l_AND_collectDataVector(
      l_databaseRowSize_ui * l_databaseColumnSize_ui);

  thrust::device_vector<long int> l_DeviceDatabase(l_databaseRowSize_ui *
                                                   l_databaseColumnSize_ui);

  // ToDo: Rename it for host_Copy.......
  thrust::host_vector<long int> l_foundedResult(l_databaseRowSize_ui *
                                                l_databaseColumnSize_ui);

  vector<long int> resultVector(l_databaseRowSize_ui * l_databaseColumnSize_ui);

  copyDataToDevice(f_dataBase_r, l_databaseRowSize_ui, l_databaseColumnSize_ui,
                   l_DeviceDatabase);

  //---------------------R-U-N----------------------
  m_firstRun_b = true;
  m_firstMethodWasOr_b = true;

  int whereIsTheTargetCharacter;
  l_collectDataVector_p = &l_AND_collectDataVector;

  for (string l_rule_str : f_selectRule) {
    whereIsTheTargetCharacter = l_rule_str.find("&");
    if (whereIsTheTargetCharacter != (-1)) {

      and_method(l_collectDataVector_p, l_AND_collectDataVector,
                 l_workDataVector, l_databaseRowSize_ui,
                 l_databaseColumnSize_ui);

      if (m_firstMethodWasOr_b)
        m_firstMethodWasOr_b = false;

      continue;
    }

    whereIsTheTargetCharacter = l_rule_str.find("|");
    if (whereIsTheTargetCharacter != (-1)) {

      continue;
    }

    /// first will be find date="2010"
    whereIsTheTargetCharacter = l_rule_str.find("=");
    if (whereIsTheTargetCharacter != (-1)) {
      equal(whereIsTheTargetCharacter, l_rule_str, l_DeviceDatabase,
            f_dataBaseHeader_v, l_collectDataVector_p, l_workDataVector,
            l_databaseRowSize_ui, l_databaseColumnSize_ui);

      continue;
    }
  }

  l_foundedResult = l_AND_collectDataVector;

  /*
   for (int x = 0; x < l_databaseRowSize_ui; x++) {
     for (int y = 0; y < l_databaseColumnSize_ui; y++) {
       printf("cuda %lu ", l_foundedResult[(x * l_databaseColumnSize_ui) + y]);
     }
     printf("\n");
   }*/

  copyDataFromDevice(l_databaseRowSize_ui, l_databaseColumnSize_ui,
                     l_foundedResult);
}

void CudaSelect::and_method(
    thrust::device_vector<long int> *f_collectDataVector_p,
    thrust::device_vector<long int> &f_AND_collectDataVector_r,
    thrust::device_vector<long int> &f_workDataVector,
    unsigned int f_rowNumber_ui, unsigned int f_columnNumber_ui) {

  thrust::host_vector<long int> nullInitVector(
      f_rowNumber_ui * f_columnNumber_ui); // by default Null vector
  f_workDataVector = nullInitVector;

  /// put collectDataVector_p contain to AND_collectDataVector_r by indirect
  // f_collectDataVector_p = &f_AND_collectDataVector_r;

  /// f_collectDataVector_p point to f_AND_collectDataVector_r !!!!!!!!!!!
  /// put the AND_collectDataVector_r contains to l_workDataVector by directly
  f_workDataVector = f_AND_collectDataVector_r;

  // similar to f_collectDataVector_p->clear();
  f_AND_collectDataVector_r = nullInitVector;
}

void CudaSelect::equal(int whereIsTheTargetCharacter, string f_SelectRule_str,
                       thrust::device_vector<long int> &dataBase_r,
                       const vector<string> &f_dataBaseHeader_v,
                       thrust::device_vector<long int> *f_collectDataVector_p,
                       thrust::device_vector<long int> &f_workDataVector,
                       unsigned int f_rowNumber_ui,
                       unsigned int f_columnNumber_ui) {

  /// date="2010"
  unsigned int l_targetColumnNumber_ui = 0;
  /// cut "=2010" part
  string column = f_SelectRule_str.substr(0, whereIsTheTargetCharacter);
  /// cut "date=" part
  string tmp_row = f_SelectRule_str.substr(whereIsTheTargetCharacter + 1,
                                           f_SelectRule_str.size());
  long int row = std::stol(tmp_row);

  /// find "date" number of column //PC side
  for (unsigned int l_it_y = 0; l_it_y < f_dataBaseHeader_v.size();
       l_it_y++) // Todo optimalize to parallel search!!
  {
    string l_column = f_dataBaseHeader_v.at(l_it_y);
    if (l_column == column) {
      l_targetColumnNumber_ui = l_it_y;
    }
  }

  if ((m_firstRun_b == true) || (m_firstMethodWasOr_b == true)) {
    /// if first time run the query, search the lines from original database
    /// else we search from workDataVector
    dim3 necessaryGridSize(f_rowNumber_ui);
    searcDataInColumn<<<1, necessaryGridSize>>>(
        thrust::raw_pointer_cast(dataBase_r.data()),
        thrust::raw_pointer_cast(f_collectDataVector_p->data()), f_rowNumber_ui,
        f_columnNumber_ui, row, l_targetColumnNumber_ui);

    // Debug point
    /*thrust::host_vector<long int> l_foundedResult(3 * 4);
    l_foundedResult= *f_collectDataVector_p;

    for (int x = 0; x < 3; x++) {
      for (int y = 0; y < 4; y++) {
        printf("result %lu ", l_foundedResult[(x * 4) + y]);
      }
      printf("\n");
    }*/

    hipDeviceSynchronize();
    m_firstRun_b = false;

  }

  else {

    dim3 necessaryGridSize(f_rowNumber_ui);
    searcDataInColumn<<<1, necessaryGridSize>>>(
        thrust::raw_pointer_cast(f_workDataVector.data()),
        thrust::raw_pointer_cast(f_collectDataVector_p->data()), f_rowNumber_ui,
        f_columnNumber_ui, row, l_targetColumnNumber_ui);

    hipDeviceSynchronize();
  }
}

vector<vector<long int>> CudaSelect::getQueryResult() const {

  return m_resultDatabase_v;
}
