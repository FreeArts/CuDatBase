#include "hip/hip_runtime.h"
#include "cuda_select.cuh"
#include <hip/hip_cooperative_groups.h>
#include <vector>

CudaSelect::CudaSelect() {

  m_firstMethodWasOr_b = true;
  m_firstRun_b = true;
  m_RunTimeMilliseconds_f = 0.0;
  m_searchRunTime_f = 0.0;

  m_necessaryBlockNumber_ui = 0;
  m_necessaryThreadNumber_ui = 0;

  m_resultDatabase_v.clear();
}

CudaSelect::~CudaSelect() {}

__global__ void searcData(long int *f_dataBase_p, long int *f_resultLines_p,
                          const unsigned long int f_databaseRowSize_ui,
                          const unsigned long int f_databaseColumnSize_ui,
                          const unsigned long int f_targetWord_ui) {

  unsigned long int rowThread = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long int columnThread = blockIdx.y * blockDim.y + threadIdx.y;

  if ((rowThread <= f_databaseRowSize_ui) &&
      (columnThread <= f_databaseColumnSize_ui)) {

    long int l_tmpWordContainer_li =
        f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + columnThread];

    if (l_tmpWordContainer_li == f_targetWord_ui) {

      for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {

        long int l_dataBaseFoundedLineContent_li =
            f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];

        f_resultLines_p[l_it_x + (rowThread * f_databaseColumnSize_ui)] =
            l_dataBaseFoundedLineContent_li;
      }
    }
  }
  // auto syncOnlyThreads= cooperative_groups::this_thread();
  // auto syncGroup = cooperative_groups::this_thread_block();
  // syncGroup.sync();
  // syncOnlyThreads.sync();
  __syncthreads();
}

__global__ void searcDataInColumn(long int *f_dataBase_p,
                                  long int *f_resultLines_p,
                                  const unsigned int f_databaseRowSize_ui,
                                  const unsigned int f_databaseColumnSize_ui,
                                  const long int f_targetWord_ui,
                                  const unsigned long int f_targetColumn) {

  int rowThread = blockIdx.x * blockDim.x + threadIdx.x;

  if (rowThread <= f_databaseRowSize_ui) {
    long int l_tmpWordContainer_li =
        f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + f_targetColumn];

    if (l_tmpWordContainer_li == f_targetWord_ui) {

      for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {

        long int l_dataBaseFoundedLineContent_li =
            f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];
        f_resultLines_p[l_it_x + (rowThread * f_databaseColumnSize_ui)] =
            l_dataBaseFoundedLineContent_li;
      }
    }
  }
  // auto syncOnlyThreads= cooperative_groups::this_thread();
  // auto syncGroup = cooperative_groups::this_thread_block();
  // syncGroup.sync();
  // syncOnlyThreads.sync();
  __syncthreads();
}

__global__ void
searcDataLessInColumn(long int *f_dataBase_p, long int *f_resultLines_p,
                      const unsigned int f_databaseRowSize_ui,
                      const unsigned int f_databaseColumnSize_ui,
                      const long int f_targetWord_ui,
                      const unsigned long int f_targetColumn) {

  int rowThread = blockIdx.x * blockDim.x + threadIdx.x;

  if (rowThread <= f_databaseRowSize_ui) {
    long int l_tmpWordContainer_li =
        f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + f_targetColumn];

    if (l_tmpWordContainer_li < f_targetWord_ui) {

      for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {

        long int l_dataBaseFoundedLineContent_li =
            f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];
        f_resultLines_p[l_it_x + (rowThread * f_databaseColumnSize_ui)] =
            l_dataBaseFoundedLineContent_li;
      }
    }
  }
  // auto syncOnlyThreads= cooperative_groups::this_thread();
  // auto syncGroup = cooperative_groups::this_thread_block();
  // syncGroup.sync();
  // syncOnlyThreads.sync();
  __syncthreads();
}

__global__ void
searcDataGreaterInColumn(long int *f_dataBase_p, long int *f_resultLines_p,
                         const unsigned int f_databaseRowSize_ui,
                         const unsigned int f_databaseColumnSize_ui,
                         const long int f_targetWord_ui,
                         const unsigned long int f_targetColumn) {

  int rowThread = blockIdx.x * blockDim.x + threadIdx.x;

  if (rowThread <= f_databaseRowSize_ui) {
    long int l_tmpWordContainer_li =
        f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + f_targetColumn];

    if (l_tmpWordContainer_li > f_targetWord_ui) {

      for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {

        long int l_dataBaseFoundedLineContent_li =
            f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];
        f_resultLines_p[l_it_x + (rowThread * f_databaseColumnSize_ui)] =
            l_dataBaseFoundedLineContent_li;
      }
    }
  }
  // auto syncOnlyThreads= cooperative_groups::this_thread();
  // auto syncGroup = cooperative_groups::this_thread_block();
  // syncGroup.sync();
  // syncOnlyThreads.sync();
  __syncthreads();
}

void CudaSelect::copyDataToDevice(
    const vector<vector<long int>> &f_dataBase_r,
    const unsigned long int f_databaseRowSize_ui,
    unsigned long int f_databaseColumnSize_ui,
    thrust::device_vector<long int> &f_DeviceDataBase_r) {

  int l_tmpDatabaseContainer_i[f_databaseRowSize_ui][f_databaseColumnSize_ui];

  unsigned int l_it_x = 0;
  unsigned int l_it_y = 0;

  for (vector<long int> vec : f_dataBase_r) {
    for (long int vector_member : vec) {
      l_tmpDatabaseContainer_i[l_it_x][l_it_y] = vector_member;
      l_it_y++;
    }
    l_it_y = 0;
    l_it_x++;
  }

  // ToDo! What??
  thrust::copy(&(l_tmpDatabaseContainer_i[0][0]),
               &(l_tmpDatabaseContainer_i[f_databaseRowSize_ui]
                                         [f_databaseColumnSize_ui]),
               f_DeviceDataBase_r.begin());
}

void CudaSelect::copyDataFromDevice(
    const unsigned long int f_databaseRowSize_ui,
    const unsigned long int f_databaseColumnSize_ui,
    const thrust::host_vector<long int> &f_resultVector) {

  vector<long int> l_tmpDatabaseContainer_v;
  for (int x = 0; x < f_databaseRowSize_ui; x++) {

    l_tmpDatabaseContainer_v.clear();
    for (int y = 0; y < f_databaseColumnSize_ui; y++) {
      long int l_tmpVectorValue =
          f_resultVector[(x * f_databaseColumnSize_ui) + y];

      l_tmpDatabaseContainer_v.push_back(l_tmpVectorValue);
    }
    m_resultDatabase_v.push_back(l_tmpDatabaseContainer_v);
  }
}

void CudaSelect::CudaRun(const vector<string> &f_selectRule,
                         const vector<vector<long int>> &f_dataBase_r,
                         const vector<string> &f_dataBaseHeader_v) {

  unsigned long int l_databaseRowSize_ui = f_dataBase_r.size();
  unsigned long int l_databaseColumnSize_ui = f_dataBaseHeader_v.size();

  calculateGridBalanceMethod(m_necessaryBlockNumber_ui,
                             m_necessaryThreadNumber_ui, l_databaseRowSize_ui);

  thrust::device_vector<long int> l_workDataVector(l_databaseRowSize_ui *
                                                   l_databaseColumnSize_ui);

  thrust::device_vector<long int> l_collectDataVector_v(
      l_databaseRowSize_ui * l_databaseColumnSize_ui);

  thrust::device_vector<long int> l_DeviceDatabase(l_databaseRowSize_ui *
                                                   l_databaseColumnSize_ui);

  // ToDo: Rename it for host_Copy.......
  thrust::host_vector<long int> l_foundedResult(l_databaseRowSize_ui *
                                                l_databaseColumnSize_ui);

  copyDataToDevice(f_dataBase_r, l_databaseRowSize_ui, l_databaseColumnSize_ui,
                   l_DeviceDatabase);

  //---------------------R-U-N----------------------
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  hipEventRecord(start);

  m_firstRun_b = true;
  m_firstMethodWasOr_b = true;

  int whereIsTheTargetCharacter;

  for (string l_rule_str : f_selectRule) {
    whereIsTheTargetCharacter = l_rule_str.find("&");
    if (whereIsTheTargetCharacter != (-1)) {

      and_method(l_collectDataVector_v, l_workDataVector, l_databaseRowSize_ui,
                 l_databaseColumnSize_ui);

      if (m_firstMethodWasOr_b)
        m_firstMethodWasOr_b = false;

      continue;
    }

    whereIsTheTargetCharacter = l_rule_str.find("|");
    if (whereIsTheTargetCharacter != (-1)) {

      continue;
    }

    /// first will be find date="2010"
    whereIsTheTargetCharacter = l_rule_str.find("=");
    if (whereIsTheTargetCharacter != (-1)) {
      find(whereIsTheTargetCharacter, l_rule_str, l_DeviceDatabase,
           f_dataBaseHeader_v, l_collectDataVector_v, l_workDataVector,
           l_databaseRowSize_ui, l_databaseColumnSize_ui, "=");

      continue;
    }

    whereIsTheTargetCharacter = l_rule_str.find("<");
    if (whereIsTheTargetCharacter != (-1)) {
      find(whereIsTheTargetCharacter, l_rule_str, l_DeviceDatabase,
           f_dataBaseHeader_v, l_collectDataVector_v, l_workDataVector,
           l_databaseRowSize_ui, l_databaseColumnSize_ui, "<");

      continue;
    }

    whereIsTheTargetCharacter = l_rule_str.find(">");
    if (whereIsTheTargetCharacter != (-1)) {
      find(whereIsTheTargetCharacter, l_rule_str, l_DeviceDatabase,
           f_dataBaseHeader_v, l_collectDataVector_v, l_workDataVector,
           l_databaseRowSize_ui, l_databaseColumnSize_ui, ">");

      continue;
    }
  }

  l_foundedResult = l_collectDataVector_v;

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&m_RunTimeMilliseconds_f, start, stop);

  /*//Only l_foundedResult value Debug:
   for (int x = 0; x < l_databaseRowSize_ui; x++) {
     for (int y = 0; y < l_databaseColumnSize_ui; y++) {
       printf("cuda %lu ", l_foundedResult[(x * l_databaseColumnSize_ui) + y]);
     }
     printf("\n");
   }*/

  copyDataFromDevice(l_databaseRowSize_ui, l_databaseColumnSize_ui,
                     l_foundedResult);
}

void CudaSelect::and_method(
    thrust::device_vector<long int> &f_collectDataVector_r,
    thrust::device_vector<long int> &f_workDataVector,
    unsigned long int f_rowNumber_ui, unsigned long int f_columnNumber_ui) {

  thrust::host_vector<long int> nullInitVector(
      f_rowNumber_ui * f_columnNumber_ui); // by default Null vector
  f_workDataVector = nullInitVector;

  /// f_collectDataVector_p point to f_collectDataVector_r !!!!!!!!!!!
  /// put the AND_collectDataVector_r contains to l_workDataVector by directly
  f_workDataVector = f_collectDataVector_r;

  // similar to f_collectDataVector_p->clear();
  f_collectDataVector_r = nullInitVector;
}

void CudaSelect::find(int whereIsTheTargetCharacter, string f_SelectRule_str,
                      thrust::device_vector<long int> &dataBase_r,
                      const vector<string> &f_dataBaseHeader_v,
                      thrust::device_vector<long int> &f_collectDataVector_r,
                      thrust::device_vector<long int> &f_workDataVector,
                      unsigned long int f_rowNumber_ui,
                      unsigned long int f_columnNumber_ui,
                      string f_mathRule_str) {

  /// date="2010"
  unsigned int l_targetColumnNumber_ui = 0;
  /// cut "=2010" part
  string column = f_SelectRule_str.substr(0, whereIsTheTargetCharacter);
  /// cut "date=" part
  string tmp_row = f_SelectRule_str.substr(whereIsTheTargetCharacter + 1,
                                           f_SelectRule_str.size());
  long int row = std::stol(tmp_row);

  /// find "date" number of column //PC side
  for (unsigned int l_it_y = 0; l_it_y < f_dataBaseHeader_v.size();
       l_it_y++) // Todo optimalize to parallel search!!
  {
    string l_column = f_dataBaseHeader_v.at(l_it_y);
    if (l_column == column) {
      l_targetColumnNumber_ui = l_it_y;
    }
  }

  if ((m_firstRun_b == true) || (m_firstMethodWasOr_b == true)) {
    /// if first time run the query, search the lines from original database
    /// else we search from workDataVector

    dim3 necessaryBlockSize(m_necessaryBlockNumber_ui);
    dim3 necessaryThreadSize(m_necessaryThreadNumber_ui);

    if (f_mathRule_str == "=") {

      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);

      searcDataInColumn<<<necessaryBlockSize, necessaryThreadSize>>>(
          thrust::raw_pointer_cast(dataBase_r.data()),
          thrust::raw_pointer_cast(f_collectDataVector_r.data()),
          f_rowNumber_ui, f_columnNumber_ui, row, l_targetColumnNumber_ui);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float l_searchTime;
      hipEventElapsedTime(&l_searchTime, start, stop);

      m_searchRunTime_f += l_searchTime;

    }

    else if (f_mathRule_str == "<") {
      searcDataLessInColumn<<<necessaryBlockSize, necessaryThreadSize>>>(
          thrust::raw_pointer_cast(dataBase_r.data()),
          thrust::raw_pointer_cast(f_collectDataVector_r.data()),
          f_rowNumber_ui, f_columnNumber_ui, row, l_targetColumnNumber_ui);
    }

    else if (f_mathRule_str == ">") {
      searcDataGreaterInColumn<<<necessaryBlockSize, necessaryThreadSize>>>(
          thrust::raw_pointer_cast(dataBase_r.data()),
          thrust::raw_pointer_cast(f_collectDataVector_r.data()),
          f_rowNumber_ui, f_columnNumber_ui, row, l_targetColumnNumber_ui);
    }

    // Debug point
    /*thrust::host_vector<long int> l_foundedResult(3 * 4);
    l_foundedResult= *f_collectDataVector_p;

    for (int x = 0; x < 3; x++) {
      for (int y = 0; y < 4; y++) {
        printf("result %lu ", l_foundedResult[(x * 4) + y]);
      }
      printf("\n");
    }*/

    hipDeviceSynchronize();
    m_firstRun_b = false;

  }

  else {

    dim3 necessaryBlockSize(m_necessaryBlockNumber_ui);
    dim3 necessaryThreadSize(m_necessaryThreadNumber_ui);

    if (f_mathRule_str == "=") {

      hipEvent_t start, stop;
      hipEventCreate(&start);
      hipEventCreate(&stop);

      hipEventRecord(start);

      searcDataInColumn<<<necessaryBlockSize, necessaryThreadSize>>>(
          thrust::raw_pointer_cast(f_workDataVector.data()),
          thrust::raw_pointer_cast(f_collectDataVector_r.data()),
          f_rowNumber_ui, f_columnNumber_ui, row, l_targetColumnNumber_ui);

      hipEventRecord(stop);
      hipEventSynchronize(stop);
      float l_searchTime;
      hipEventElapsedTime(&l_searchTime, start, stop);

      m_searchRunTime_f += l_searchTime;

    }

    else if (f_mathRule_str == "<") {
      searcDataLessInColumn<<<necessaryBlockSize, necessaryThreadSize>>>(
          thrust::raw_pointer_cast(f_workDataVector.data()),
          thrust::raw_pointer_cast(f_collectDataVector_r.data()),
          f_rowNumber_ui, f_columnNumber_ui, row, l_targetColumnNumber_ui);
    }

    else if (f_mathRule_str == ">") {
      searcDataGreaterInColumn<<<necessaryBlockSize, necessaryThreadSize>>>(
          thrust::raw_pointer_cast(f_workDataVector.data()),
          thrust::raw_pointer_cast(f_collectDataVector_r.data()),
          f_rowNumber_ui, f_columnNumber_ui, row, l_targetColumnNumber_ui);
    }

    hipDeviceSynchronize();
  }
}

void CudaSelect::calculateGridFillMethod(
    unsigned long int &f_necessaryBlockNumber_r,
    unsigned long int &f_necessaryThreadNumber_r,
    const unsigned long int f_rowNumber_ui) {

  f_necessaryBlockNumber_r = f_rowNumber_ui / 500;

  if (f_necessaryBlockNumber_r <= 0) {
    f_necessaryBlockNumber_r = 1;
    f_necessaryThreadNumber_r = f_rowNumber_ui;
  }

  else {
    // maximum Thread/block
    f_necessaryThreadNumber_r = 500;
  }
}

void CudaSelect::calculateGridBalanceMethod(
    unsigned long int &f_necessaryBlockNumber_r,
    unsigned long int &f_necessaryThreadNumber_r,
    const unsigned long int f_rowNumber_ui) {

  // maximum 500 thread/block
  f_necessaryBlockNumber_r = f_rowNumber_ui / 500;

  if (f_necessaryBlockNumber_r <= 0) {
    f_necessaryBlockNumber_r = 1;
    f_necessaryThreadNumber_r = f_rowNumber_ui;
  }

  else {
    unsigned long int l_remainder_ui =
        f_rowNumber_ui % f_necessaryBlockNumber_r;
    if (l_remainder_ui == 0) {
      f_necessaryThreadNumber_r = f_rowNumber_ui / f_necessaryBlockNumber_r;
    } else {
      f_necessaryThreadNumber_r =
          (f_rowNumber_ui / f_necessaryBlockNumber_r) + 1;
    }
  }
}

vector<vector<long int>> CudaSelect::getQueryResult() const {

  return m_resultDatabase_v;
}

float CudaSelect::getRuntimeValue() const { return m_RunTimeMilliseconds_f; }

float CudaSelect::getSearchtimeValue() const { return m_searchRunTime_f; }
