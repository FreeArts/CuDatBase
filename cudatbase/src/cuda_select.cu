#include "hip/hip_runtime.h"

#include "cuda_select.cuh"
#include <hip/hip_cooperative_groups.h>
#include <vector>

//------Real Functions----

CudaSelect::CudaSelect() {
  // copyDataToDevice();
}

CudaSelect::~CudaSelect() {}

__device__ int stepper = 0;
__global__ void searcData(long int *f_dataBase_p, long int *f_resultLines_p,
                          const unsigned int f_databaseRowSize_ui,
                          const unsigned int f_databaseColumnSize_ui,
                          const unsigned int f_targetWord_ui) {

  int rowThread = blockIdx.x * blockDim.x + threadIdx.x;
  int columnThread = blockIdx.y * blockDim.y + threadIdx.y;

  long int temp =
      f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + columnThread];
  if (temp == f_targetWord_ui) {
    // printf("Founded threadPair: %i %i\n",rowThread,columnThread);
    for (int l_it_x = 0; l_it_x < f_databaseColumnSize_ui; l_it_x++) {
      long int l_dataBaseFoundedLineContent_li =
          f_dataBase_p[(rowThread * f_databaseColumnSize_ui) + l_it_x];
      f_resultLines_p[l_it_x + stepper] = l_dataBaseFoundedLineContent_li;
      // printf("Array %d \n",
      // f_resultLines_p[(f_databaseColumnSize_ui)+l_it_x]);
    }
    atomicAdd(&stepper, f_databaseColumnSize_ui);
  }

  auto syncGroup = cooperative_groups::this_thread_block();
  syncGroup.sync();
  //__syncthreads();
}

void CudaSelect::copyDataToDevice(
    const vector<vector<long int>> &f_dataBase_r,
    const unsigned int f_databaseRowSize_ui,
    unsigned int f_databaseColumnSize_ui,
    thrust::device_vector<long int> &f_DeviceDataBase_r) {
  int l_tmpDatabaseContainer_i[f_databaseRowSize_ui][f_databaseColumnSize_ui];

  unsigned int l_it_x = 0;
  unsigned int l_it_y = 0;

  for (vector<long int> vec : f_dataBase_r) {
    for (long int vector_member : vec) {
      l_tmpDatabaseContainer_i[l_it_x][l_it_y] = vector_member;
      l_it_y++;
    }
    l_it_y = 0;
    l_it_x++;
  }

  printf("\n");

  // What??
  thrust::copy(&(l_tmpDatabaseContainer_i[0][0]),
               &(l_tmpDatabaseContainer_i[f_databaseRowSize_ui]
                                         [f_databaseColumnSize_ui]),
               f_DeviceDataBase_r.begin());
}

void CudaSelect::CudaRun(const vector<string> &f_selectRule,
                         const vector<vector<long int>> &f_dataBase_r,
                         const vector<string> &f_dataBaseHeader_v) {

  unsigned int l_databaseRowSize_ui = f_dataBase_r.size();
  unsigned int l_databaseColumnSize_ui = f_dataBaseHeader_v.size();

  // thrust::device_vector<long int> *l_collectDataVector_p(NULL);
  // thrust::device_vector<long int>
  // l_workDataVector(l_databaseRowSize_ui*l_databaseColumnSize_ui);

  // thrust::device_vector<long int>
  // l_AND_collectDataVector(l_databaseRowSize_ui*l_databaseColumnSize_ui);
  // thrust::device_vector<long int>
  // l_OR_collectDataVector(l_databaseRowSize_ui*l_databaseColumnSize_ui);

  thrust::device_vector<long int> l_DeviceDatabase(l_databaseRowSize_ui *
                                                   l_databaseColumnSize_ui);
  thrust::device_vector<long int> l_DeviceResult(l_databaseRowSize_ui *
                                                 l_databaseColumnSize_ui);
  thrust::host_vector<long int> l_foundedResult(l_databaseRowSize_ui *
                                                l_databaseColumnSize_ui);

  copyDataToDevice(f_dataBase_r, l_databaseRowSize_ui, l_databaseColumnSize_ui,
                   l_DeviceDatabase);

  dim3 grid(l_databaseRowSize_ui, l_databaseColumnSize_ui);
  searcData<<<grid, 1>>>(thrust::raw_pointer_cast(l_DeviceDatabase.data()),
                         thrust::raw_pointer_cast(l_DeviceResult.data()),
                         l_databaseRowSize_ui, l_databaseColumnSize_ui, 2010);
  hipDeviceSynchronize();

  l_foundedResult = l_DeviceResult;

  for (int x = 0; x < l_databaseRowSize_ui; x++) {
    for (int y = 0; y < l_databaseColumnSize_ui; y++) {
      printf("Result %lu ", l_foundedResult[(x * l_databaseColumnSize_ui) + y]);
    }
    printf("\n");
  }
}

void CudaSelect::or_method(
    thrust::device_vector<long int> *f_collectDataVector_p,
    thrust::device_vector<long int> &f_OR_collectDataVector_r) {}

void CudaSelect::and_method(
    thrust::device_vector<long int> *f_collectDataVector_p,
    const thrust::device_vector<long int> &f_OR_collectDataVector_r,
    thrust::device_vector<long int> &f_AND_collectDataVector_r,
    thrust::device_vector<long int> &f_workDataVector) {}

void CudaSelect::or_and_merge(
    const thrust::device_vector<long int> *f_collectDataVector_p,
    const thrust::device_vector<long int> &f_OR_collectDataVector_r,
    thrust::device_vector<long int> &f_AND_collectDataVector_r) {}

void CudaSelect::equal(int input, string f_SelectRule_str,
                       const thrust::device_vector<long int> &dataBase_r,
                       thrust::device_vector<long int> *f_collectDataVector_p,
                       thrust::device_vector<long int> &f_workDataVector,
                       bool &firstRun, unsigned int f_columnNumber_ui) {}
//-----------------------------------------------------

void testVector() {

  // https://www.geeksforgeeks.org/convert-string-char-array-cpp/
  printf("TestFunction");
  thrust::host_vector<char *> host_vector;
  vector<string> simple_vector;
  vector<char *> simple_result_vector;
  thrust::device_vector<char *> device_vector;

  simple_vector.push_back("Hello");
  simple_vector.push_back("vilag");
  simple_vector.push_back("Szia");

  for (int i = 0; i < simple_vector.size(); i++) {
    // char* s = (char*)"Hello";
    // device_vector.push_back(s);

    std::string str = simple_vector.at(i);
    char *s = const_cast<char *>(str.c_str());
    // strcpy
    host_vector.push_back(s);

    // simple_result_vector.at(i) = device_vector[i];
    // host_vector = device_vector;
  }

  device_vector = host_vector;

  cout << "host_v " << (simple_vector[0]) << endl;
  cout << "host_v " << (host_vector[0]) << endl;
  cout << "host_v " << (device_vector[0]) << endl;

  cout << "host_v " << &(simple_vector[0]) << endl;
  cout << "host_v " << &(host_vector[0]) << endl;
  cout << "host_v " << &(device_vector[0]) << endl;
}

// example so test--------------------
__global__ void addKernel(int *c, const int *a, const int *b, int size) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    c[i] = a[i] + b[i];
  }
}

void addWithCuda(int *c, const int *a, const int *b, int size) {
  int *dev_a = nullptr;
  int *dev_b = nullptr;
  int *dev_c = nullptr;

  // Allocate GPU buffers for three vectors (two input, one output)
  hipMalloc((void **)&dev_c, size * sizeof(int));
  hipMalloc((void **)&dev_a, size * sizeof(int));
  hipMalloc((void **)&dev_b, size * sizeof(int));

  // Copy input vectors from host memory to GPU buffers.
  hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

  // Launch a kernel on the GPU with one thread for each element.
  // 2 is number of computational blocks and (size + 1) / 2 is a number of
  // threads in a block
  addKernel<<<2, (size + 1) / 2>>>(dev_c, dev_a, dev_b, size);

  // hipDeviceSynchronize waits for the kernel to finish, and returns
  // any errors encountered during the launch.
  hipDeviceSynchronize();

  // Copy output vector from GPU buffer to host memory.
  hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

  hipFree(dev_c);
  hipFree(dev_a);
  hipFree(dev_b);
}

void callExample(int a[5], int b[5], int c[5]) {
  // const int a[5] = {  1,  2,  3,  4,  5 };
  // const int b[5] = { 10, 20, 30, 40, 50 };
  // int c[5] = { 0 };

  addWithCuda(c, a, b, 5);

  hipDeviceReset();
}
//------------------------------------------
void testCuda(void) {
  // test_kernel <<<1, 1>>> ();
  printf("Hello, world!");
  work();
}
//-----------------
/*
/**
 * CUDA kernel that computes reciprocal values for a given vector
 */
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
  unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < vectorSize)
    data[idx] = 1.0 / data[idx];
}

/**
 * Host function that copies the data and launches the work on GPU
 */
float *gpuReciprocal(float *data, unsigned size) {
  float *rc = new float[size];
  float *gpuData;

  CUDA_CHECK_RETURN(hipMalloc((void **)&gpuData, sizeof(float) * size));
  CUDA_CHECK_RETURN(
      hipMemcpy(gpuData, data, sizeof(float) * size, hipMemcpyHostToDevice));

  static const int BLOCK_SIZE = 256;
  const int blockCount = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  reciprocalKernel<<<blockCount, BLOCK_SIZE>>>(gpuData, size);

  CUDA_CHECK_RETURN(
      hipMemcpy(rc, gpuData, sizeof(float) * size, hipMemcpyDeviceToHost));
  CUDA_CHECK_RETURN(hipFree(gpuData));
  return rc;
}

float *cpuReciprocal(float *data, unsigned size) {
  float *rc = new float[size];
  for (unsigned cnt = 0; cnt < size; ++cnt)
    rc[cnt] = 1.0 / data[cnt];
  return rc;
}

void initialize(float *data, unsigned size) {
  for (unsigned i = 0; i < size; ++i)
    data[i] = .5 * (i + 1);
}

int work(void) {
  static const int WORK_SIZE = 65530;
  float *data = new float[WORK_SIZE];

  initialize(data, WORK_SIZE);

  float *recCpu = cpuReciprocal(data, WORK_SIZE);
  float *recGpu = gpuReciprocal(data, WORK_SIZE);
  float cpuSum = std::accumulate(recCpu, recCpu + WORK_SIZE, 0.0);
  float gpuSum = std::accumulate(recGpu, recGpu + WORK_SIZE, 0.0);

  /* Verify the results */
  std::cout << "gpuSum = " << gpuSum << " cpuSum = " << cpuSum << std::endl;

  /* Free memory */
  delete[] data;
  delete[] recCpu;
  delete[] recGpu;

  return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */

static void CheckCudaErrorAux(const char *file, unsigned line,
                              const char *statement, hipError_t err) {
  if (err == hipSuccess)
    return;
  std::cerr << statement << " returned " << hipGetErrorString(err) << "("
            << err << ") at " << file << ":" << line << std::endl;
  exit(1);
}
